#ifndef MEASURE_CU
#define MEASURE_CU

#include <chrono>

struct Measure {
  std::chrono::time_point<std::chrono::high_resolution_clock> mStart;
  std::chrono::time_point<std::chrono::high_resolution_clock> mEnd;

  Measure() {} 

  void fromNow() {
    mStart = std::chrono::high_resolution_clock::now();
  }

  void untilNow() {
    mEnd = std::chrono::high_resolution_clock::now();
  }

  float millis() {
    return nanos() / 1000000.0; 
  }

  float micros() {
    return nanos() / 1000.0; 
  }

  float nanos() {
    return std::chrono::duration_cast<std::chrono::nanoseconds>(mEnd - mStart).count() * 1.0;
  }
};

struct CudaMeasure {
  hipEvent_t mStart;
  hipEvent_t mEnd;
  float elapsedMs;

  CudaMeasure(): elapsedMs(0) {
    hipEventCreate(&mStart);
    hipEventCreate(&mEnd);
  }

  ~CudaMeasure() {
    hipEventDestroy(mStart);
    hipEventDestroy(mEnd);
  }

  void fromNow() {
    hipEventRecord(mStart, 0);
  }

  void untilNow() {
    hipEventRecord(mEnd, 0);
    hipEventSynchronize(mEnd);

    float elapsedTime = 0;
    hipEventElapsedTime(&elapsedTime, mStart, mEnd);

    elapsedMs = elapsedTime;
  }

  float millis() {
    return elapsedMs;
  }

  float micros() {
    return  (elapsedMs * 1000.0);
  }

  float nanos() {
    return  (elapsedMs * 1000000.00);
  }

  float value() {
    return elapsedMs;
  }

};

#endif
